using namespace std;

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <fstream>
//#include "cuPrintf.cu"

#define B0 -2.647866f
#define B1 -0.374927f
#define B2 0.061601f
#define B3 -0.001511f

const long MAX_THREADS = 512;
const long MAX_BLOCK= 65535;

extern "C" void botrix_index (float *tempday_h, float* precday_h, int n, float* output);
extern "C" void init ();
void checkCUDAError(const char* msg);

__global__ void calculate_index(float* tempday, float* precday, int n)
{
  int id;
  float x, y;
  id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
  x=tempday[id];
  y=precday[id];
  if (id<n)
  {
    if (x!=-9999 && y!=-9999)
    {
      if (y>=4 && x<40)
      {
	if (x<12)
	  x=12;
	if (x>32)
	  x=32;
	tempday[id]=powf(M_E,(B0+(B1*y)+(B2*y*x)+(B3*y*(x*x)))) / (1+powf(M_E,(B0+(B1*y)+(B2*y*x)+(B3*y*(x*x)))));
    
      }
      else
	tempday[id]=0;
    }
    else
      tempday[id]=-9999;
  }
}

void init()
{
  hipSetDevice(0);
}

void botrix_index(float *tempday_h, float* precday_h, int n, float* output)
{
  float *tempday_d, *precday_d;
  long n_threads;
  int n_blocks;
  int dim_blocks;
  dim3 dim_grid;

  //selezione device da utilizzare
//  cudaSetDevice(0);

  //settaggio parametri
  n_threads=n;
  if (n_threads>MAX_BLOCK*MAX_BLOCK*MAX_THREADS)
  {
    printf("Troppi threads!\n");
    exit(0);
  }
  dim_blocks=MAX_THREADS;
  n_blocks=n_threads/MAX_THREADS+(n_threads%MAX_THREADS==0?0:1);
  if (n_blocks<=MAX_BLOCK)
  {
    dim_grid.x=n_blocks;
    dim_grid.y=1;
    dim_grid.z=1;
  }
  else
  {
    dim_grid.x=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.y=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.z=1;
  }
  printf("Numero threads per blocco: %d\n",dim_blocks);
  printf("Dimensioni grid: x %d,  y %d\n",dim_grid.x,dim_grid.y);

  //stampa input
//  cout << "tempday\n";
//  for (int i=0; i<n; i++)
//  {
//    cout<<tempday_h[i]<<" ";
//  }
//  cout << endl;
//  cout << "precday\n";
//  for (int i=0; i<n; i++)
//  {
//    cout<<precday_h[i]<<" ";
//  }
//  cout << endl;

  //allocazione
  hipMalloc((void**) &precday_d, n*sizeof(float));
  hipMalloc((void**) &tempday_d, n*sizeof(float));
  checkCUDAError("Allocazione");

  //trasferimento su device
  hipMemcpy(tempday_d,tempday_h, sizeof(float)*n, hipMemcpyHostToDevice);
  hipMemcpy(precday_d,precday_h, sizeof(float)*n, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
 
  //lancio kernel
  //cudaPrintfInit();
  calculate_index <<< dim_grid, dim_blocks >>>(tempday_d, precday_d, n);
  //cudaPrintfDisplay(stdout, true);
  checkCUDAError("Kernel");

  //trasferimento da device
  hipMemcpy(output,tempday_d, sizeof(float)*n, hipMemcpyDeviceToHost);
  checkCUDAError("Trasferimento da device");

//  cout << "output\n";
//  for (int i=0; i<n; i++)
//  {
//    cout<<output[i]<<" ";
//  }
//  cout << endl;

  //deallocazione
  hipFree(precday_d);
  hipFree(tempday_d);
}

int main(int argc, char **argv)
{
  fstream tempday, precday;
  int n;
  float *tempday_h, *precday_h, *output;

  //lettura parametri
  if (argc<3)
  {
    printf("./a.out tempday.txt precday.txt lunghezza\n");
    exit(0);
  }
  tempday.open(argv[1],ios::in);
  precday.open(argv[2],ios::in);
  sscanf(argv[3],"%d",&n);

  //allocazione
  precday_h = (float*) malloc (n*sizeof(float));
  tempday_h = (float*) malloc (n*sizeof(float));
  output = (float*) malloc (n*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (int i=0; i<n; i++)
  {
    tempday >> tempday_h[i];
    precday >> precday_h[i];
  }

  botrix_index(tempday_h,precday_h,n,output);

  //stampa
  cout << "Risultato botrite:\n";
  for (int i=0; i<n; i++)
  {
    cout<<output[i]<<" ";
  }

  //deallocazione
  free(precday_h);
  free(tempday_h);
  return 0;
}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}


