using namespace std;

#include "debug_cuda.h"
#include <iostream>
#include <cstdlib>

int main()
{
  int r,c, n;
  float *vect_h, *vect_d, *matr_h, *matr_d;
  

  n=100;
  r=20;
  c=5;

  vect_h =new float [n];
  matr_h =new float [r*c];
  hipMalloc((void**) &vect_d, n*sizeof(float));
  hipMalloc((void**) &matr_d, r*c*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (int i=0; i<n; i++)
    vect_h[i]=(float) i/(i+2)*100;
  for (int i=0; i<r*c; i++)
    matr_h[i]=(float) i/(i+2)*100;
  
  //stampa
  for (int i=0; i<n; i++)
  {
    cout << vect_h[i] << " ";
  }
  cout << endl;
  
  //stampa
  for (int i=0; i<r; i++)
  {
    for (int j=0; j<c; j++)
      cout << matr_h[i*c+j] << " ";
    cout << endl;
  }

  cout<<endl;
  cout<<endl;

  //trasferimento su device
  hipMemcpy(vect_d,vect_h, sizeof(float)*n, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
  hipMemcpy(matr_d,matr_h, sizeof(float)*r*c, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
  
  //stampa
  print_vector(vect_d,n);
  print_matrix(matr_d,r,c);
  return 0;
}
