using namespace std;

#include "debug_cuda.h"
#include <iostream>
#include <cstdlib>
#include <iomanip>

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
      cerr << "Cuda error: "<<msg<<": "<<hipGetErrorString(err)<<".\n";
      exit(1);
    }                         
}

void print_vector(float *data, int n)
{
  float *temp;
  temp=new float [n];
  hipMemcpy(temp,data, sizeof(float)*n, hipMemcpyDeviceToHost);
  checkCUDAError("DEBUG Download data");
  for (int i=0; i<n; i++)
   cout << temp[i] << " ";
  cout << endl;
  delete [] temp;
}

void print_vector(int *data, int n)
{
  int *temp;
  temp=new int [n];
  hipMemcpy(temp,data, sizeof(int)*n, hipMemcpyDeviceToHost);
  checkCUDAError("DEBUG Download data");
  for (int i=0; i<n; i++)
    cout << temp[i] << " ";
  cout << endl;
  delete [] temp;
}

void print_matrix(float *data, int r, int c)
{
  float *temp;
  temp=new float [r*c];
  hipMemcpy(temp,data, sizeof(float)*r*c, hipMemcpyDeviceToHost);
  checkCUDAError("DEBUG Download data");
  for (int i=0; i<r; i++)
  {
    for (int j=0; j<c; j++)
      cout << setw(10) << temp [i*c+j];
    cout << endl;
  }
  delete [] temp;
}
 
void print_matrix(int *data, int r, int c)
{
  int *temp;
  temp=new int [r*c];
  hipMemcpy(temp,data, sizeof(int)*r*c, hipMemcpyDeviceToHost);
  checkCUDAError("DEBUG Download data");
  for (int i=0; i<r; i++)
  {
    for (int j=0; j<c; j++)
      cout << setw(15) << temp [i*c+j];
    cout << endl;
  }
  delete [] temp;
}
 
