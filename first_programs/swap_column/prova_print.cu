#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include<hip/hip_runtime.h>

__global__ void testKernel(int val)
{
  cuPrintf("Value is: %d\n", threadIdx.x);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
  {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }                         
}

int main()
{
  dim3 blocks;
  blocks.x=65535;
  blocks.y=65535;
  blocks.z=1;
  cudaPrintfInit();
  testKernel<<< blocks, 512 >>>(10);
  checkCUDAError("Kernel");
  cudaPrintfDisplay(stdout, true);
  cudaPrintfEnd();
  return 0;
}
