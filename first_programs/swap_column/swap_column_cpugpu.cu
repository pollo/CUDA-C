#include "hip/hip_runtime.h"
using namespace std;

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "cuPrintf.cu"

const long MAX_THREADS = 512;
const long MAX_BLOCK= 65535;

void checkCUDAError(const char* msg);

void swap(float &a, float &b)
{
  float t;
  t=a;
  a=b;
  b=t;
}

__global__ void swap(float* dati, long n, long c)
{
  long id;
  float t;
  id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
//  cuPrintf("Id: %ld\n", id);  
  if (id<n/2)
  {
//isorgente=id/(c/2)
//jsorgente=id%(c/2)
//idsorgente=isorgente*c+jsorgente
//idsorgenteesteso=(id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)
//idtarget=idsorgente+(c-(idsorgete%c)*2-1)
//idtargetesteso=((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/+(c-( ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/%c)*2-1)
//    cuPrintf("isorgente: %ld\n",id/(c/2));
//    cuPrintf("jsorgente: %ld\n",id%(c/2));
//    cuPrintf("idsorgente: %ld\n",(id/(c/2))/*is*/*c+ (id%(c/2))/*js*/);
//    cuPrintf("idtarget: %ld\n", ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/+(c-( ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/%c)*2-1));
    t=dati[(id/(c/2))*c+ (id%(c/2))];
    dati[(id/(c/2))*c+ (id%(c/2))]= dati[((id/(c/2))*c+ (id%(c/2)))+(c-( ((id/(c/2))*c+ (id%(c/2)))%c)*2-1)];
    dati[((id/(c/2))*c+ (id%(c/2)))+(c-( ((id/(c/2))*c+ (id%(c/2)))%c)*2-1)]=t;
  }
}

int main(int argc, char **argv)
{
  timeval time;
  double t1, t2, t3, t4;
  long r,c;
  float *dati_h, *dati_d, *dati;
  long int n_threads, n_blocks;
  int dim_blocks;
  dim3 dim_grid;
  bool check; 

  //settaggio parametri
  if (argc<3)
  {
    printf("./a.out r c\n");
    exit(0);
  }
  sscanf(argv[1],"%ld",&r);
  sscanf(argv[2],"%ld",&c);
  n_threads=r*(c/2);
  if (n_threads>MAX_BLOCK*MAX_BLOCK*MAX_THREADS)
  {
    printf("Troppi threads!\n");
    exit(0);
  }
  dim_blocks=MAX_THREADS;
  n_blocks=n_threads/MAX_THREADS+(n_threads%MAX_THREADS==0?0:1);
  if (n_blocks<=MAX_BLOCK)
  {
    dim_grid.x=n_blocks;
    dim_grid.y=1;
    dim_grid.z=1;
  }
  else
  {
    dim_grid.x=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.y=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.z=1;
  }
  printf("Numero threads per blocco: %d\n",dim_blocks);
  printf("Dimensioni grid: x %d,  y %d\n",dim_grid.x,dim_grid.y);

  //inizio cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t1=time.tv_sec+(time.tv_usec/1000000.0);

  //allocazione
  dati_h = (float*) malloc (r*c*sizeof(float));
  hipMalloc((void**) &dati_d, r*c*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (long i=0; i<r*c; i++)
    dati_h[i]=i;

   //trasferimento su device
  hipMemcpy(dati_d,dati_h, sizeof(float)*r*c, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
 
  //inizio cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t3=time.tv_sec+(time.tv_usec/1000000.0);

  //lancio kernel
  cudaPrintfInit();
  swap <<< dim_grid, dim_blocks >>>(dati_d, (r*c), c);
  cudaPrintfDisplay(stdout, true);
  checkCUDAError("Kernel");

  //stoppa cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t4=time.tv_sec+(time.tv_usec/1000000.0);

  //trasferimento da device
  hipMemcpy(dati_h,dati_d, sizeof(float)*r*c, hipMemcpyDeviceToHost);
  checkCUDAError("Trasferimento da device");

  //stoppa cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t2=time.tv_sec+(time.tv_usec/1000000.0);
  printf("Tempo impiegato dal kernel: %f\n",t4-t3);
  printf("Tempo impiegato gpu: %f\n",t2-t1);
  
  //inizio cronometro
  gettimeofday(&time, NULL);
  t1=time.tv_sec+(time.tv_usec/1000000.0);
  
  //allocazione
  dati = (float*) malloc (r*c*sizeof(float));
  
  //inizializzazione
  for (long i=0; i<r*c; i++)
    dati[i]=i;
  
  //inizio cronometro
  gettimeofday(&time, NULL);
  t3=time.tv_sec+(time.tv_usec/1000000.0);

  //swap
  for (long i=0; i<r; i++)
    for (long j=0; j<c/2; j++)
      swap(dati[i*c+j],dati[(i+1)*c-j-1]);

  //stoppa cronometro
  gettimeofday(&time, NULL);
  t4=time.tv_sec+(time.tv_usec/1000000.0);

  //stoppa cronometro
  gettimeofday(&time, NULL);
  t2=time.tv_sec+(time.tv_usec/1000000.0);
  printf("Tempo impiegato processo: %f\n",t4-t3);
  printf("Tempo impiegato cpu: %f\n",t2-t1);  

  //check
  check=true;
  for (long i=0; i<r*c; i++)
    check=(dati_h[i]==dati[i]);
  if (check)
    printf("Tutto ok!\n");
  else
    printf("Errore\n");

  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}


