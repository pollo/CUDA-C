#include "hip/hip_runtime.h"
using namespace std;

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include "cuPrintf.cu"

const long MAX_THREADS = 512;
const long MAX_BLOCK= 65535;

void checkCUDAError(const char* msg);

__global__ void swap(float* dati, long n, long c)
{
  long id;
  float t;
  id=(blockIdx.y*gridDim.x+blockIdx.x)*blockDim.x+threadIdx.x;
//  cuPrintf("Id: %ld\n", id);  
  if (id<n/2)
  {
//isorgente=id/(c/2)
//jsorgente=id%(c/2)
//idsorgente=isorgente*c+jsorgente
//idsorgenteesteso=(id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)
//idtarget=idsorgente+(c-(idsorgete%c)*2-1)
//idtargetesteso=((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/+(c-( ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/%c)*2-1)
//    cuPrintf("isorgente: %ld\n",id/(c/2));
//    cuPrintf("jsorgente: %ld\n",id%(c/2));
//    cuPrintf("idsorgente: %ld\n",(id/(c/2))/*is*/*c+ (id%(c/2))/*js*/);
//    cuPrintf("idtarget: %ld\n", ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/+(c-( ((id/(c/2))/*is*/*c+ (id%(c/2))/*js*/)/*ids*/%c)*2-1));
    t=dati[(id/(c/2))*c+ (id%(c/2))];
    dati[(id/(c/2))*c+ (id%(c/2))]= dati[((id/(c/2))*c+ (id%(c/2)))+(c-( ((id/(c/2))*c+ (id%(c/2)))%c)*2-1)];
    dati[((id/(c/2))*c+ (id%(c/2)))+(c-( ((id/(c/2))*c+ (id%(c/2)))%c)*2-1)]=t;
  }
}

int main(int argc, char **argv)
{
  timeval time;
  double t1, t2;
  long r,c;
  float *dati_h, *dati_d;
  long int n_threads, n_blocks;
  int dim_blocks;
  dim3 dim_grid;
 
  //settaggio parametri
  if (argc<3)
  {
    printf("./a.out r c\n");
    exit(0);
  }
  sscanf(argv[1],"%ld",&r);
  sscanf(argv[2],"%ld",&c);
//   cout << r <<endl<< c<<endl;
  n_threads=r*(c/2);
  if (n_threads>MAX_BLOCK*MAX_BLOCK*MAX_THREADS)
  {
    printf("Troppi threads!\n");
    exit(0);
  }
  dim_blocks=MAX_THREADS;
  n_blocks=n_threads/MAX_THREADS+(n_threads%MAX_THREADS==0?0:1);

//  cout << "Numero di threads " << n_threads << endl;    
//  cout << "Numero di blocchi " << n_blocks << endl;

  if (n_blocks<=MAX_BLOCK)
  {
    dim_grid.x=n_blocks;
    dim_grid.y=1;
    dim_grid.z=1;
  }
  else
  {
    dim_grid.x=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.y=(unsigned int) ceil(sqrt(n_blocks));
    dim_grid.z=1;
  }
  printf("Numero threads per blocco: %d\n",dim_blocks);
  printf("Dimensioni grid: x %d,  y %d\n",dim_grid.x,dim_grid.y);

  //allocazione
  dati_h = (float*) malloc (r*c*sizeof(float));
  hipMalloc((void**) &dati_d, r*c*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (long i=0; i<r*c; i++)
    dati_h[i]=i;
  
  //stampa
//  for (long i=0; i<r; i++)
//  {
//    for (long j=0; j<c; j++)
//      printf("%7.2f",dati_h[i*c+j]);
//    printf("\n");
//  }

  //trasferimento su device
  hipMemcpy(dati_d,dati_h, sizeof(float)*r*c, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");

  //inizio cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t1=time.tv_sec+(time.tv_usec/1000000.0);
 
  //lancio kernel
  cudaPrintfInit();
  swap <<< dim_grid, dim_blocks >>>(dati_d, (r*c), c);
  cudaPrintfDisplay(stdout, true);
  checkCUDAError("Kernel");

  //stoppa cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t2=time.tv_sec+(time.tv_usec/1000000.0);
  printf("Tempo impiegato: %f\n",t2-t1);

  //trasferimento da device
  hipMemcpy(dati_h,dati_d, sizeof(float)*r*c, hipMemcpyDeviceToHost);
  checkCUDAError("Trasferimento da device");

  //stampa    
//  printf("\n");
//  for (long i=0; i<r; i++)
//  {
//    for (long j=0; j<c; j++)
//      printf("%7.2f",dati_h[i*c+j]);
//    printf("\n");
//  }
  

  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}


