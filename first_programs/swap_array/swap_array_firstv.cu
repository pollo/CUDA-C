#include "hip/hip_runtime.h"
/*
  Swap the elements of a vector: the first with the last and so on...
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <sys/time.h>

void checkCUDAError(const char* msg);

__global__ void rebalta (float *dati,  long n)
{
  long id;
  long t;
  id=blockIdx.x*blockDim.x+threadIdx.x;
  if (id<n/2)
  {
    t=dati[n-id-1];
    dati[n-id-1]=dati[id];
    dati[id]=t; 
  }
}

int main(int argc, char **argv)
{
  long n;
  long i;
  timeval time;
  double t1, t2;
  float *dati_h;
  float *dati_d;
  long blocksize, nblocks;
  srand(1);

  //inizio cronometro
  gettimeofday(&time, NULL);
  t1=time.tv_sec+(time.tv_usec/1000000.0);
 
  //settaggio parametri
  if (argc<2)
  {
    printf("./a.out n\n");
    exit(0);
  }
  sscanf(argv[1],"%ld",&n);
  blocksize=512;
  nblocks=(n/2)/blocksize + ((n/2)%blocksize == 0?0:1);
  printf ("numero blocchi %ld\n", nblocks);
  printf ("numero threads %ld\n", blocksize);

  //allocazione
  dati_h=(float *) malloc (n*sizeof(float));
  hipMalloc((void**) &dati_d, n*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (i=0; i<n; i++)
    dati_h[i]=(rand()%100000)/1000.0;

//  for (i=0; i<n; i++)
//    printf("%f ",dati_h[i]);
//  printf("\n");
  
  //trasferimento su device
  hipMemcpy(dati_d,dati_h, sizeof(float)*n, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
  
  //lancio kernel
  rebalta <<< nblocks, blocksize >>> (dati_d, n);
  checkCUDAError("Kernel");

  //trasferimento da device
  hipMemcpy(dati_h,dati_d, sizeof(float)*n, hipMemcpyDeviceToHost);
  checkCUDAError("Trasferimento da device");
  
  //stoppa cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t2=time.tv_sec+(time.tv_usec/1000000.0);
  printf("Tempo impiegato: %f\n",t2-t1);

//  for (i=0; i<n; i++)
//    printf("%f ",dati_h[i]);
//  printf("\n");
  return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
