#include "hip/hip_runtime.h"
/*
  Swap the elements of a vector: the first with the last and so on...
*/

#include<hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<sys/time.h>

void checkCUDAError(const char* msg);

__global__ void rebalta (float *dati, int n)
{
  int id;
  int t;
  id=blockIdx.x*blockDim.x+threadIdx.x;
  t=dati[n-id-1];
  dati[n-id-1]=dati[id];
  dati[id]=t; 
}

__global__ void last_rebalta (float *dati, int n, int blockid, int blockdim)
{
  int id;
  int t;
  id=blockid*blockdim+threadIdx.x;
  t=dati[n-id-1];
  dati[n-id-1]=dati[id];
  dati[id]=t; 
}



int main(int argc, char **argv)
{
  int n;
  int i;
  timeval time;
  double t1, t2;
  float *dati_h;
  float *dati_d;
   int blocksize, nblocks;
  srand(1);

  //inizio cronometro
  gettimeofday(&time, NULL);
  t1=time.tv_sec+(time.tv_usec/1000000.0);
 
  //settaggio parametri
  if (argc<2)
  {
    printf("./a.out n\n");
    exit(0);
  }
  sscanf(argv[1],"%d",&n);
  blocksize=512;
  nblocks=(n/2)/blocksize;
  printf ("prima volta\n");
  printf ("numero blocchi %d\n", nblocks);
  printf ("numero threads %d\n", blocksize);
  printf ("seconda volta\n");
  printf ("numero blocchi %d\n", 1);
  printf ("numero threads %d\n", (n/2)%blocksize);
  

  //allocazione
  dati_h=(float *) malloc (n*sizeof(float));
  hipMalloc((void**) &dati_d, n*sizeof(float));
  checkCUDAError("Allocazione");

  //inizializzazione
  for (i=0; i<n; i++)
    dati_h[i]=(rand()%100000)/1000.0;

//  for (i=0; i<n; i++)
//    printf("%f ",dati_h[i]);
//  printf("\n");
  
  //trasferimento su device
  hipMemcpy(dati_d,dati_h, sizeof(float)*n, hipMemcpyHostToDevice);
  checkCUDAError("Trasferimento su device");
  
  //lancio kernel
  if (nblocks>0 && blocksize>0)
    rebalta <<< nblocks, blocksize >>> (dati_d, n);
  checkCUDAError("Kernel");
  if ((n/2)%blocksize>0)
    last_rebalta <<< 1, (n/2)%blocksize >>> (dati_d, n, nblocks, blocksize);
  checkCUDAError("Kernel");

  //trasferimento da device
  hipMemcpy(dati_h,dati_d, sizeof(float)*n, hipMemcpyDeviceToHost);
  checkCUDAError("Trasferimento da device");
  
  //stoppa cronometro
  hipDeviceSynchronize();
  gettimeofday(&time, NULL);
  t2=time.tv_sec+(time.tv_usec/1000000.0);
  printf("Tempo impiegato: %f\n",t2-t1);

//  for (i=0; i<n; i++)
//    printf("%f ",dati_h[i]);
//  printf("\n");
  return 0;
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
  {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }                         
}
