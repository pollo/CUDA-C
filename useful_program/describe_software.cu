
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void)
{
  int driverVersion, runtimeVersion;
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("driver version %d runtime version %d\n",
	 driverVersion, runtimeVersion);
  return 0;
}
