#include <stdio.h>
#include <hip/hip_runtime.h>


int main()
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("Numero di device disponibili   %d\n\n\n",deviceCount);
  int device;
  for (device = 0; device < deviceCount; ++device) 
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Proprieta device %d:\n\n",device);
    int canMapHostMemory=deviceProp.canMapHostMemory;
    printf("Device can map host memory with hipHostAlloc/hipHostGetDevicePointer. %d\n", canMapHostMemory);
    int	clockRate=deviceProp.clockRate;
    printf("Clock frequency in kilohertz. %d\n", clockRate);
    int	computeMode=deviceProp.computeMode;
    printf ("Compute mode (See hipComputeMode). %d\n",computeMode);
    int deviceOverlap=deviceProp.deviceOverlap;
    printf("Device can concurrently copy memory and execute a kernel. %d\n",deviceOverlap);
    int integrated=deviceProp.integrated;
    printf("Device is integrated as opposed to discrete. %d\n",integrated);
    int kernelExecTimeoutEnabled=deviceProp.kernelExecTimeoutEnabled;
    printf("Specified whether there is a run time limit on kernels. %d\n",kernelExecTimeoutEnabled);
    int major=deviceProp.major;
    printf("Major compute capability. %d\n",major);
    int	maxGridSize[3];
    maxGridSize[0]=deviceProp.maxGridSize[0];
    maxGridSize[1]=deviceProp.maxGridSize[1];
    maxGridSize[2]=deviceProp.maxGridSize[2];
    printf("Maximum size of each dimension of a grid. %d %d %d\n",maxGridSize[0],maxGridSize[1],maxGridSize[2]);
    int	maxThreadsDim[3];
    maxThreadsDim[0]=deviceProp.maxThreadsDim[0];
    maxThreadsDim[1]=deviceProp.maxThreadsDim[1];
    maxThreadsDim[2]=deviceProp.maxThreadsDim[2];
    printf("Maximum size of each dimension of a block. %d %d %d\n",maxThreadsDim[0],maxThreadsDim[1],maxThreadsDim[2]);
    int maxThreadsPerBlock=deviceProp.maxThreadsPerBlock;
    printf("Maximum number of threads per block. %d\n",maxThreadsPerBlock);
    size_t memPitch=deviceProp.memPitch;
    printf("Maximum pitch in bytes allowed by memory copies. %lu\n", (unsigned long) memPitch);
    int minor=deviceProp.minor;
    printf("Minor compute capability. %d\n",minor);
    int multiProcessorCount=deviceProp.multiProcessorCount;
    printf("Number of multiprocessors on device. %d\n",multiProcessorCount);
    char name[256];
    for (int i=0; i<256; i++)
      name[i]=deviceProp.name[i];
    printf ("ASCII string identifying device. %s\n",name);
    int	regsPerBlock=deviceProp.regsPerBlock;
    printf ("32-bit registers available per block %d\n",regsPerBlock);
    size_t sharedMemPerBlock=deviceProp.sharedMemPerBlock;
    printf ("Shared memory available per block in bytes. %lu\n", (unsigned long) sharedMemPerBlock);
    size_t textureAlignment=deviceProp.textureAlignment;
    printf ("Alignment requirement for textures. %lu\n", (unsigned long) textureAlignment);
    size_t totalConstMem=deviceProp.totalConstMem;
    printf ("Constant memory available on device in bytes. %lu\n", (unsigned long) totalConstMem);
    size_t totalGlobalMem=deviceProp.totalGlobalMem;
    printf ("Global memory available on device in bytes. %lu\n", (unsigned long) totalGlobalMem);
    int	warpSize=deviceProp.warpSize;
    printf ("Warp size in threads. %d\n", warpSize);
    printf("\n");
  } 
}
